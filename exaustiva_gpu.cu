#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <map>
#include <random>
#include <iomanip>
#include <chrono>
#include <fstream>

using namespace std;

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>

struct filme{
    int id;
    int inicio;
    int fim;
    int categoria;
};

// Functor que aplica a busca exaustiva
struct exaustiva_GPU
{
    int N;
    int C;
    int *cat_limites;
    filme *filmes;

    exaustiva_GPU(int _N, int _C, int *_cat_limites, filme *_filmes) 
    : N(_N), C(_C), cat_limites(_cat_limites), filmes(_filmes) {}

    __device__ int operator()(int i) const
    {
        int local_limites[50];  

        // int tempoSemTela = 24;
        int maratonaAtual = 0;

        filme last_added;
        bool has_added = false;

        // Inicializa limites locais
        for (int j = 0; j < 50; j++) {
            local_limites[j] = 0;
        }

        for (int j = 0; j < N; j++){
            if ((i & (1 << j))){

                // Verifica se a categoria do filme já alcançou o limite
                if (local_limites[filmes[j].categoria - 1] >= cat_limites[filmes[j].categoria - 1]) {
                    return -1;
                }

                if (!has_added){
                    has_added = true;
                }else{
                    int fimlast_added = last_added.fim > last_added.inicio ? last_added.fim : last_added.fim + 24;
                    int fimNovoFilme = filmes[j].fim > filmes[j].inicio ? filmes[j].fim : filmes[j].fim + 24;

                    if (!((filmes[j].inicio >= fimlast_added) || (fimNovoFilme <= last_added.inicio))) {
                        return -1;
                    }
                }

                // Calcula o tempo de tela do filme
                int tempoFilme = 0;

                if (filmes[j].fim >= filmes[j].inicio){
                    tempoFilme = filmes[j].fim - filmes[j].inicio;
                } else {
                    tempoFilme = filmes[j].fim + 24 - filmes[j].inicio;
                }

                // Verifica se há tempo de tela disponível
                if (tempoSemTela < tempoFilme) {
                    return -1;
                }

                // Decrementa conforme adiciona filme
                tempoSemTela -= tempoFilme;

                maratonaAtual++;
                local_limites[filmes[j].categoria - 1]++;
                
                last_added = filmes[j];
            }
        }
        
        // Retorna a quantidade de filmes na maratona atual
        return maratonaAtual;
    }
};

int main() {

    auto start_time = std::chrono::high_resolution_clock::now();

    int N = 0; // Número de filmes
    int C = 0;  // Número de categorias

    cin >> N >> C;

    // Ler os dados do arquivo de entrada
    vector<int> limites;
    vector<filme> filmes, maratona; 

    // Captura o número de filmes para cada categoria
    int cat_limites;
    for (int i = 0; i < C; i++){
        cin >> cat_limites;
        limites.push_back(cat_limites);
    }

    // Captura os filmes
    for (int i = 0; i < N; i++){
        int inicio, fim, categoria;
        cin >> inicio >> fim >> categoria;
        filmes.push_back({i, inicio, fim, categoria});
    }
    
    // Ordena os filmes por hora de início crescente
    sort(filmes.begin(), filmes.end(), [](const filme& a, const filme& b){return a.inicio < b.inicio;});

    // --------------------------------------- Carregamento na CPU ---------------------------------------

    // Carregar os dados do arquivo de entrada na memória da GPU
    thrust::device_vector<filme> filmes_gpu(filmes);
    thrust::device_vector<int> categorias_gpu(limites);

    // Cria um device vector para os resultados
    thrust::device_vector<int> resultados(pow(2, N));

    exaustiva_GPU functor(N, C, thrust::raw_pointer_cast(categorias_gpu.data()), thrust::raw_pointer_cast(filmes_gpu.data()));

    // Aplica o functor
    thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(pow(2, N)), resultados.begin(), functor);

    // --------------------------------------------- Output ---------------------------------------------

    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    double time_taken = duration.count()/1000.0;

    int totalDeFilmes = *thrust::max_element(resultados.begin(), resultados.end());

    cout << "\nFilmes assistidos na Exaustiva com GPU: " << totalDeFilmes << endl;

    // Escreve o resultado em um arquivo de saída
    ofstream myfile;
    myfile.open ("output_exaustiva_gpu.txt", ios::app);
    myfile << N << " " << C << " " << time_taken << " " << totalDeFilmes << "\n";
    myfile.close();

    return 0;

}